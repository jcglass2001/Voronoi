#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "./ppmFile.c" 

#define MAX_VALUE 1000
#define MAX_WIDTH 1680
#define MAX_HEIGHT 1050

struct Point{
    float r,g,b;
    float x,y;
};

hipError_t cuda_ret;

__global__ void kernel( Point *p, unsigned char *ptr, int sites ) {
    // Map from blockIdx to pixel position
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    int offset = x + (y * blockDim.x*gridDim.x);

    // Now find the closest site and color the current pixel accordingly
    // Hint: In a for loop, measure the Euclidean Distance between the current
    //       pixel and each site. Find the shortest.


    int minDistance = sqrt(pow(MAX_WIDTH,2) + pow(MAX_HEIGHT,2));//LONG_MAX;
    int closest_pt_idx = -1;
    //find closest point
    for(unsigned int i = 0; i < sites; i++){
      int distance = sqrt(pow(p[i].x - x, 2) + pow(p[i].y - y, 2));
      if(distance < minDistance){
        minDistance = distance;
        closest_pt_idx = i;
      } 
    }
    //set color
    ptr[offset*3+0] = p[closest_pt_idx].r;
    ptr[offset*3+1] = p[closest_pt_idx].g;
    ptr[offset*3+2] = p[closest_pt_idx].b;


}

int getDivisor(int numerator)
{
    int divisor = 31;
    while((numerator % divisor) != 0) {divisor--;}
    return divisor;
}

int main(int argc, char **argv) {

    if(argc < 4) {
        printf("usage: voronoi <width> <height> <sites>");
        exit(0);
    }

    // 1. Obtain width, height of the Voronoi Diagram image and the number of sites from command line ------
    // Ensure they are within the pre-set max caps 

    int w = atoi(argv[1]), h = atoi(argv[2]), points = atoi(argv[3]);
    w = (w <= MAX_WIDTH)? w : MAX_WIDTH;
    h = (h <= MAX_HEIGHT)? h : MAX_HEIGHT;
    points = (points <= MAX_VALUE)? points : MAX_VALUE;

    // 2. Allocate device memory for the Voronoi Diagram image ----------------------------------------------
    // Use "Julia Set" for reference
    unsigned char *d_copy;
    cuda_ret = hipMalloc((void** )&d_copy, sizeof(unsigned char*) * (w*h*3));
    if(cuda_ret != hipSuccess) fprintf(stderr, "%s\n", "Unable to allocate device memory (Voronoi Diagram Image)");

    // 3. Sites --------------------------------------------------------------------------------------------
    // 3.1. Allocate device memory for the sites 
    // The type of each site is a structure named "Point" defined at the top of this program
    struct Point *d_Sites;
    cuda_ret = hipMalloc((void**) &d_Sites, sizeof(struct Point) * points);
    if(cuda_ret != hipSuccess) fprintf(stderr, "%s\n", "Unable to allocate device memory (Sites)");

    // 3.2. Allocate host memory for the the sites. 
    // Then for each site, 
    //  - assign a random value say rand()%255+50 to its r, g, and b elemment respectively
    //  - assign a random value say rand()%w to its x, where w is the width of the Voronoi Diagram image
    //  - assign a random value say rand()%h to its y, where h is the height of the Voronoi Diagram image
    struct Point *h_Sites = (struct Point*)malloc(sizeof(struct Point) * points);
    for(unsigned int i = 0; i < points; i++){

      h_Sites[i].r = rand()%255 + 50;
      h_Sites[i].g = rand()%255 + 50;
      h_Sites[i].b = rand()%255 + 50;

      h_Sites[i].x = rand()%w;
      h_Sites[i].y = rand()%h;
    }

    // 3.3. Copy all sites from host memory to device memory
    printf("Copying all sites from host to device...");
    cuda_ret = hipMemcpy(d_Sites, h_Sites, sizeof(struct Point) * points, hipMemcpyHostToDevice);

    // Launch kernel ----------------------------------------------------------------------------------------
    // The pre-defined function "getDivisor" comes convenient. 
    // When it's called with w, getDivisor(w), it generates the largest number (<=31) that divides w 
    // When it's called with h, getDivisor(h), it generates the largest number (<=31) that divides h
    // These two results become demisions of a block 
    int wGrid = getDivisor(w);
    int hGrid = getDivisor(h);

    dim3 dimGrid(w/wGrid, h/hGrid);
    dim3 dimBlock(wGrid, hGrid);

    printf("\ndimGrid: (%d, %d)", w/wGrid, h/hGrid);
    printf("\ndimBlock: (%d, %d)", wGrid, hGrid);
    printf("\nResolution: %dx%d", w, h);
    printf("\nSites: %d\n\n", points);

    // Replace the following line with your code ...
    kernel<<<dimGrid,dimBlock>>>(d_Sites, d_copy, points);



    Image *outImage;
    outImage = ImageCreate(w,h);
    ImageClear(outImage,0,0,0);
    
    // Copy device variables to host ----------------------------------------
    // Use "Julia Set" for reference
    cuda_ret = hipMemcpy(outImage -> data, d_copy, w*h*3, hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) fprintf(stderr, "%s\n", "Unable to copy device variables to host (Sites)");


    // Convert image to the ppm formt and free the host memory.----------------------------------------
    // Use "Julia Set" for reference
    const char* outFile = "out.ppm";
    ImageWrite(outImage, outFile);

    // Free device memory
    free(h_Sites);
    hipFree(d_Sites); hipFree(d_copy);
    free(outImage->data);
    
}

